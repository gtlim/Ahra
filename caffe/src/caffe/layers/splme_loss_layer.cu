#include "hip/hip_runtime.h"
/**
 * @brief 
 *  this is ranking loss with structure preserving LME
 *  with basic multi-prototype clustering added
 *  structure preserving constraint.
 *  implemented by gtlim 2015.9.25
 **/


/**
 * bottom[0] similarity or distance measure. btw instances
 *  [ batch ] x [ num_cate * num_proto ]
 * bottom[1] similarity or distance measure. btw prototypes
 * bottom[2] label of instance
 * bottom[3] multi_label table of given label
 * [category_label] [ num_proto ] 
 * bottom[4] structure preserving constraint graph
 **/

#include <algorithm>
#include <cfloat>
#include <cmath>
#include <vector>

#include "caffe/layers/splme_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

// define for input information table
#define DIM 3   // dimension of table
#define OFF 2   // offset of given prototypes
#define NUMP 1  // number of prototypes
#define INDEX 0 // cateogry label of given prototypes

// define for sorter 
#define DIMS 2   // dimension of sorter

namespace caffe {

//precompute minimum of all category.
template <typename Dtype>
__global__ void preComputeGPU(const int nthreads,
 const Dtype* s_value, const Dtype* table, const Dtype* graph,
 Dtype* sorter, const int MaxNum) {
 const int outter_dim = nthreads;
 CUDA_KERNEL_LOOP(index, nthreads) {
  const int num_proto = table[index*DIM+NUMP];
  const int offset    = table[index*DIM+OFF];
  Dtype max_val = s_value[index*outter_dim+offset ]; 
  int max_idx = offset;
  bool init = false;
  for(int i = 0 ; i < num_proto; i++) {
    //initial value of connected component
    if( graph[index*MaxNum +i ] && !init ) {
     max_val = s_value[index*outter_dim + offset+i]; 
     max_idx = offset+i;
     init = true;
    }
    if( init && graph[index*MaxNum + i] && max_val  < s_value[index*outter_dim + offset + i] ) {
     max_val = s_value[index*outter_dim + offset + i];
     max_idx = offset + i;
    } 
  }
  sorter[index*DIMS] = max_val;
  sorter[index*DIMS+1] = max_idx;
 }
}

//computing structure preserving constraint loss
template <typename Dtype>
__global__ void spConstraintLossForwardGPU(const int nthreads,
    const Dtype* s_val, const Dtype* bottom_label, Dtype* sloss, 
    const Dtype* graph,const Dtype* table ,const Dtype* sorter,
    const int outter_dim,const int MaxNum,const Dtype margin) {
 CUDA_KERNEL_LOOP(index, nthreads) {
  const int n = index / outter_dim; //row of instance.
  //const int label = static_cast<int>(bottom_label[n]);
  const int label = n;
  const int label_index = table[label*DIM+INDEX];  //category index.
  const int position = index%outter_dim;
  const int offset = table[position*DIM+OFF];
  if(table[position*DIM+INDEX] == label_index && graph[label*MaxNum + position - offset] )
    continue; //except nearest neighbor
  const Dtype prob = max( Dtype(0), 
    margin + sorter[label*DIMS] - s_val[label*outter_dim+position]);
  sloss[index] = (prob > 0 ) ? prob : 0;
 }
}

//computing category loss pair-wise Distance
template <typename Dtype>
__global__ void spLmeLossPair_Distance_ForwardGPU(const int nthreads,
    const Dtype* score, const Dtype* label,const Dtype* table,
    Dtype* loss, const int outter_dim,const Dtype margin,const Dtype inner_margin) {
 CUDA_KERNEL_LOOP(index, nthreads) {
  const int n = index / outter_dim; //row of score.
  const int position = index%outter_dim;
  const int label_value = static_cast<int>(label[n]);
  const int label_index = table[label_value*DIM+INDEX]; //category index.
  const int offset = n*outter_dim + label_value;
  if( offset == index ) {  
   continue; 
  } else if( table[position*DIM+INDEX] != label_index) {
  // pair_wise version of loss 
   loss[index] += max( Dtype(0),
      margin + score[offset] - score[index] );
  } else if( table[position*DIM+INDEX] == label_index && inner_margin > 0) {
  // pair_wise for multi_labels 
   loss[index] += max( Dtype(0),
      inner_margin + score[offset] - score[index] );  
  }
 }
}

//computing category loss pair-wise Distance
template <typename Dtype>
__global__ void spLmeLossUnary_Distance_ForwardGPU(const int nthreads,
    const Dtype* score, const Dtype* label,const Dtype* table,
    Dtype* loss, const int outter_dim,const Dtype margin,const Dtype inner_margin) {
 CUDA_KERNEL_LOOP(index, nthreads) {
  const int n = index / outter_dim; //row of score.
  const int position = index%outter_dim;
  const int label_value = static_cast<int>(label[n]);
  const int label_index = table[label_value*DIM+INDEX]; //category index.
  const int offset = n*outter_dim + label_value;
  if( offset == index ) {  
   loss[index] += max( Dtype(0),
      score[index] - margin );
  } else if( table[position*DIM+INDEX] != label_index) {
   loss[index] += max( Dtype(0),
      inner_margin - score[index]);
  } else if( table[position*DIM+INDEX] == label_index && inner_margin > 0) {
   loss[index] += max( Dtype(0),
      inner_margin - score[index]);
  }
 }
}

template <typename Dtype>
void spLmeLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Dtype closs,wloss,sloss;
  wloss = Dtype(0.);
  //computing category loss 
  const Dtype* score = bottom[0]->gpu_data(); //score of L2 norm or similarity
  const Dtype* bottom_label = bottom[2]->gpu_data(); //own label of instance( which is from clustering)
  const Dtype* table_ = bottom[3]->gpu_data();  // information table
  const int outter_loop = bottom[0]->num(); //batch_size
  const int outter_dim = bottom[0]->count()/outter_loop; // total number of prototypes
 
  const int nthreads = bottom[0]->count(); // [ batch_size x number of prototypes ]
  Blob<Dtype> c_loss(bottom[0]->shape());
  Dtype* loss_data = c_loss.mutable_gpu_data();
  //select version of terms.
  //computing main loss ( large margin embedding with multi-prototypes)
  if( version == "pair" ) {
   spLmeLossPair_Distance_ForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
       CAFFE_CUDA_NUM_THREADS>>>(nthreads, score , bottom_label, table_, 
        loss_data, outter_dim, margin,inner_margin);  
  } else if( version == "unary") {
   spLmeLossUnary_Distance_ForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
       CAFFE_CUDA_NUM_THREADS>>>(nthreads, score , bottom_label, table_, 
        loss_data, outter_dim, margin,inner_margin);  
  } else if( version == "verify" ) {
   caffe_gpu_set(nthreads,Dtype(0.),loss_data);
  } else {
    LOG(FATAL) << this->type() 
               << " wrong version " << version;
  }
  caffe_gpu_asum(nthreads, loss_data, &closs);
  wloss+=closs;
  sorter_gpu.Reshape(outter_dim,DIMS,1,1);
  const Dtype* graph = bottom[4]->gpu_data();
  const Dtype* s_val = bottom[1]->gpu_data();
  const int MaxNum = bottom[4]->count()/outter_dim;
  //computing structure preserving constraints for category
  //precomputing minimum or maximun of all Uc
  preComputeGPU<Dtype><<<CAFFE_GET_BLOCKS(outter_dim),CAFFE_CUDA_NUM_THREADS>>>
  (outter_dim, s_val, table_ , graph, sorter_gpu.mutable_gpu_data(), MaxNum);
  Blob<Dtype> s_loss(bottom[1]->shape());
  caffe_gpu_set(s_loss.count(),Dtype(0.),s_loss.mutable_gpu_data());
  Dtype* sloss_data = s_loss.mutable_gpu_data();
  const Dtype* sorter_val = sorter_gpu.gpu_data();
  const int nthread = bottom[1]->count(); // [ batch_size x number of prototypes ]
  //computing structure preserving constraint loss
  spConstraintLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthread),
    CAFFE_CUDA_NUM_THREADS>>>(nthread,s_val,bottom_label,sloss_data,
     graph,table_ ,sorter_val,outter_dim,MaxNum, sp_margin);
  caffe_gpu_asum(nthread, sloss_data, &sloss);
  if( verbose ) {
    
   top[1]->mutable_cpu_data()[0] = wloss/outter_loop;
   top[2]->mutable_cpu_data()[0] = lambda*sloss/bottom[1]->num();
  }
  wloss /= outter_loop;
  wloss+= lambda*(sloss/bottom[1]->num());
  top[0]->mutable_cpu_data()[0] = wloss;
}

template <typename Dtype>
__global__ void spLmeLossPair_Distance_BackwardGPU( const int nthreads, const Dtype* score,
   const Dtype* label, const Dtype* table, Dtype* bottom_diff,
   const int outter_dim, const Dtype margin, const Dtype inner_margin) {
 unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
 if(ix < nthreads) { 
  const int n = ix; //row of score.
  const int label_value = static_cast<int>(label[n]);
  const int index_ = table[label_value*DIM+INDEX]; //category index.
  //check multi labels
  for(int j = 0 ; j < outter_dim ; j++) {
   //if( table[j*DIM+INDEX] == index_ ) continue;
   // pair_wise version of loss 
   if( (ix*outter_dim+label_value) == (ix*outter_dim+j)) {
    continue;
   } else if( table[j*DIM+INDEX] != index_) {
    const Dtype prob = max( Dtype(0),
      margin + score[ix*outter_dim + label_value] - score[ix*outter_dim + j] );
    if( prob > 0){
     bottom_diff[ix*outter_dim + j] -= 1;
     bottom_diff[ix*outter_dim + label_value] += 1;  
    }
   } else if( table[j*DIM+INDEX] == index_ && inner_margin > 0 ) {
    const Dtype prob = max( Dtype(0),
      inner_margin + score[ix*outter_dim + label_value] - score[ix*outter_dim + j] );
    if( prob > 0){
     bottom_diff[ix*outter_dim + j] -= 1;
     bottom_diff[ix*outter_dim + label_value] += 1;  
    }
   }
  }
 }
}


template <typename Dtype>
__global__ void spLmeLossUnary_Distance_BackwardGPU( const int nthreads, const Dtype* score,
   const Dtype* label, const Dtype* table, Dtype* bottom_diff,
   const int outter_dim, const Dtype margin, const Dtype inner_margin) {
 unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
 if(ix < nthreads) { 
  const int n = ix; //row of score.
  const int label_value = static_cast<int>(label[n]);
  const int index_ = table[label_value*DIM+INDEX]; //category index.
  //check multi labels
  for(int j = 0 ; j < outter_dim ; j++) {
   //if( table[j*DIM+INDEX] == index_ ) continue;
   // pair_wise version of loss 
   if( (ix*outter_dim+label_value) == (ix*outter_dim+j)) {
    const Dtype prob = max( Dtype(0),
      score[ix*outter_dim + label_value] - margin);
    if( prob > 0){
     bottom_diff[ix*outter_dim + j] += 1;
    } 
   } else if( table[j*DIM+INDEX] != index_) {
    const Dtype prob = max( Dtype(0),
      inner_margin - score[ix*outter_dim + j] );
    if( prob > 0){
     bottom_diff[ix*outter_dim + j] -= 1;
    } 
   } else if( table[j*DIM+INDEX] == index_ && inner_margin > 0 ) {
    const Dtype prob = max( Dtype(0),
      inner_margin - score[ix*outter_dim + j] );
    if( prob > 0){
     bottom_diff[ix*outter_dim + j] -= 1;
    } 
   }
  }
 }
}


template <typename Dtype>
void spLmeLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[2] || propagate_down[3] || propagate_down[4]) {
   LOG(FATAL) << this->type() 
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
   const int num = bottom[0]->num();
   const Dtype scale = top[0]->cpu_diff()[0]/num;
   const Dtype* score = bottom[0]->gpu_data();
   const Dtype* bottom_label = bottom[2]->gpu_data();
   const Dtype* table_  = bottom[3]->gpu_data(); // information table
   const int outter_loop = bottom[0]->num(); //batch_size;
   const int outter_dim  = bottom[0]->count()/outter_loop; // total number of prototypes;

   const int nthreads = bottom[0]->num(); 
   //initialize
   Dtype* bottom_diff = bottom[0]->mutable_gpu_diff(); 
   caffe_gpu_set(bottom[0]->count(),Dtype(0.),bottom[0]->mutable_gpu_diff());
   //select the version of loss
   if( version == "pair" ) {
    spLmeLossPair_Distance_BackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, score , bottom_label, table_, 
         bottom_diff, outter_dim, margin,inner_margin);  
    caffe_gpu_scal(bottom[0]->count(), scale , bottom[0]->mutable_gpu_diff());
   } else if( version == "unary" ) {
    spLmeLossUnary_Distance_BackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, score , bottom_label, table_, 
         bottom_diff, outter_dim, margin,inner_margin);  
    caffe_gpu_scal(bottom[0]->count(), scale , bottom[0]->mutable_gpu_diff());

   } else if( version == "verify") {
   
   } else {
    LOG(FATAL) << this->type() 
               << " wrong version " << version;
   }
  }
  //back propagate with respect to structure preserving constraint.
  if (propagate_down[1]) {
   const int num = bottom[1]->num();
   const Dtype scale = lambda*top[0]->cpu_diff()[0]/num;
   const Dtype* graph  = bottom[4]->cpu_data();
   const Dtype* s_value  = bottom[1]->cpu_data();
   const Dtype* bottom_label = bottom[2]->cpu_data();
   const Dtype* table_  = bottom[3]->cpu_data(); 
   //const int outter_loop = bottom[0]->num(); //batch_size;
   const int outter_loop = bottom[0]->count()/bottom[0]->num(); //batch_size;
   //const int outter_dim  = bottom[0]->count()/outter_loop; // total number of prototypes
   const int outter_dim  = outter_loop; // total number of prototypes
   const int MaxNum = bottom[4]->count()/outter_dim;  // Max number of prototypes
   const Dtype* sorter_val = sorter_gpu.cpu_data();
   Dtype* bottom_diff = bottom[1]->mutable_cpu_diff();
   caffe_set(bottom[1]->count(), Dtype(0), bottom_diff);

   for( int i = 0 ; i < outter_loop ; i++) {
    //const int label = static_cast<int>(bottom_label[i]);
    const int label = i;
    if( label == -1 ) continue;
    const int label_index = table_[label*DIM+INDEX]; //category index
    for(int j = 0; j < outter_dim ; ++j) { 
     if( table_[j*DIM+INDEX] == label_index && graph[label*MaxNum + j - int(table_[j*DIM+OFF])] ) 
        continue; //except nearest neighbors.
     const Dtype prob = std::max( Dtype(0),
       sp_margin + sorter_val[label*DIMS] - s_value[label*outter_dim + j] );
     if( prob > 0 ) {
      bottom_diff[ label*outter_dim + int(sorter_val[label*DIMS+1])] += 1;
      bottom_diff[ label*outter_dim + j] -= 1;
     }
    }
   }
   caffe_scal(bottom[1]->count(), scale, bottom_diff);
  }
}
INSTANTIATE_LAYER_GPU_FUNCS(spLmeLossLayer);

}  // namespace caffe
