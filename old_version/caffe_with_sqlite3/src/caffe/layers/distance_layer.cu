#include "hip/hip_runtime.h"
/*
 *
 * @brief this is for generic verison of 
 *  computing distance between instance and proto types
 *  in case of CEDL, there is two types of proto
 *  one is for category proto 
 *   and the other is for attribute 
 *  good luck implemented by gtlim 2015.9.24
 *
 *  please synchronize cpp and cu version at the same time.
 *  this layer is highly optimizied by using cuda implementation
 * 
 */

#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/distance_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
__global__ void prepBackwardGPU(const int nthreads,
const Dtype* top_diff, Dtype*scales,
const int row, const int col,const bool mode ) {
 CUDA_KERNEL_LOOP(index,nthreads){ 
  for( int iy = 0 ; iy < col ; iy++){
     Dtype scale = ( mode ) ? top_diff[index*col+iy] : top_diff[iy*row+index];
     scales[index] += scale;
  }
 }
}

template <typename Dtype>
__global__ void DistanceLayerBackwardGPU(const int nthreads,
const Dtype* x_data, const Dtype* prepdata, const Dtype* top_diff, 
const Dtype*scales, Dtype* bottom_diff, const int dim) {
 CUDA_KERNEL_LOOP(index,nthreads){
  const int ix = index/dim; // axis of M
  bottom_diff[index] = Dtype(2.)*(scales[ix]*x_data[index] - prepdata[index]);
 }
}

// computing summation.
template <typename Dtype>
__global__ void prepSummationGPU(const int nthreads,
const Dtype* in, Dtype* out, const int dim) {
 CUDA_KERNEL_LOOP(index,nthreads){ 
  out[index] = 0;
  for( int iy = 0 ; iy < dim ; iy++){
     out[index]+= in[index*dim+iy]; 
  }
 }
}

// computing average.
template <typename Dtype>
__global__ void bsxfunPlusGPU(const int nthreads,
const Dtype* x,const Dtype* y, Dtype* out, const int dim,const int mode) {
 CUDA_KERNEL_LOOP(index,nthreads){ 
  int pos;
  if( mode == 0 ) {
   // row in this case
   pos = index/dim;
  } else if( mode == 1) {
   // column in this case
   pos = index%dim;
  }
  out[index] = x[pos] + y[index]; 
 }
}


template <typename Dtype>
void DistanceLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
   const vector<Blob<Dtype>*>& top) {

  const Dtype* img_feat = bottom[0]->gpu_data(); 
  const Dtype* proto = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int dim = bottom[0]->count()/bottom[0]->num(); //assume that bottom[0] & bottom[1] dimension is same.
  CHECK_EQ(bottom[0]->count()/bottom[0]->num(),bottom[1]->count()/bottom[1]->num()) <<
   "W Dimension and U Dimension is not computable";
  //computing Distance between proto and img_feat

  Blob<Dtype> Y(bottom[0]->shape());
  Blob<Dtype> Y2(bottom[0]->num(),1,1,1);
  Blob<Dtype> X(bottom[1]->shape());
  Blob<Dtype> X2(bottom[1]->num(),1,1,1);
  Blob<Dtype> Z(M_,N_,1,1);
  Blob<Dtype> M(M_,N_,1,1);
  Dtype* y = Y.mutable_gpu_data();  
  Dtype* x = X.mutable_gpu_data();  
  Dtype* z = Z.mutable_gpu_data();  

  caffe_gpu_mul(bottom[0]->count(),img_feat,img_feat,y);
  prepSummationGPU<Dtype><<< CAFFE_GET_BLOCKS(bottom[0]->num()),
      CAFFE_CUDA_NUM_THREADS >>>(bottom[0]->num(),Y.gpu_data(),Y2.mutable_gpu_data(),dim);
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
                          img_feat, proto, (Dtype)0., z);   
  caffe_gpu_scal(Z.count(),Dtype(-2),Z.mutable_gpu_data());
  bsxfunPlusGPU<Dtype><<< CAFFE_GET_BLOCKS(M.count()),
      CAFFE_CUDA_NUM_THREADS >>>(M.count(),Y2.gpu_data(),Z.gpu_data(),M.mutable_gpu_data(),bottom[1]->num(),0);
  caffe_gpu_mul(bottom[1]->count(),proto,proto,x);
  prepSummationGPU<Dtype><<< CAFFE_GET_BLOCKS(bottom[1]->num()),
      CAFFE_CUDA_NUM_THREADS >>>(bottom[1]->num(),X.gpu_data(),X2.mutable_gpu_data(),dim);
  bsxfunPlusGPU<Dtype><<< CAFFE_GET_BLOCKS(M.count()),
      CAFFE_CUDA_NUM_THREADS >>>(M.count(),X2.gpu_data(),M.gpu_data(),top_data,bottom[1]->num(),1);
  
}

template <typename Dtype>
void DistanceLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  const Dtype* img_feat = bottom[0]->gpu_data();
  const Dtype* proto = bottom[1]->gpu_data();
  if(propagate_down[0] ) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    Blob<Dtype> prep(M_,K_,1,1),scales(M_,1,1,1);
    Blob<Dtype> X(M_,1,1,1);
    Dtype* pdata = prep.mutable_gpu_data();
    Dtype* sdata = scales.mutable_gpu_data();
    caffe_gpu_set(prep.count(),Dtype(0.),pdata);
    caffe_gpu_set(scales.count(),Dtype(0.),sdata);
    caffe_gpu_set(bottom[0]->count(),Dtype(0.),bottom_diff);
    // backprop to img_feat
    // compute corresponding values
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
                          top_diff, proto, (Dtype)0., pdata);   
    // pre-compute scales
    prepBackwardGPU<Dtype><<< CAFFE_GET_BLOCKS(M_),
      CAFFE_CUDA_NUM_THREADS >>>(M_,top_diff,sdata,M_, N_,1);
    DistanceLayerBackwardGPU<Dtype><<< CAFFE_GET_BLOCKS(M_*K_),
      CAFFE_CUDA_NUM_THREADS >>>(M_*K_,img_feat, pdata,top_diff,sdata,bottom_diff,K_);
  }
  if(propagate_down[1] ) {   
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[1]->mutable_gpu_diff();
    Blob<Dtype> prep(N_,K_,1,1),scales(N_,1,1,1);
    Blob<Dtype> X(N_,1,1,1);
    Dtype* pdata = prep.mutable_gpu_data();
    Dtype* sdata = scales.mutable_gpu_data();
    caffe_gpu_set(prep.count(),Dtype(0.),pdata);
    caffe_gpu_set(scales.count(),Dtype(0.),sdata);
    caffe_gpu_set(bottom[1]->count(),Dtype(0.),bottom_diff);
    //backprop to proto
    // compute corresponding values
    caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
                          top_diff, img_feat, (Dtype)0., pdata);   
    // pre-compute scales
    prepBackwardGPU<Dtype><<< CAFFE_GET_BLOCKS(N_),
      CAFFE_CUDA_NUM_THREADS >>>(N_,top_diff,sdata,N_,M_,0);
    DistanceLayerBackwardGPU<Dtype><<< CAFFE_GET_BLOCKS(N_*K_),
      CAFFE_CUDA_NUM_THREADS >>>(N_*K_,proto,pdata,top_diff,sdata,bottom_diff,K_);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DistanceLayer);
}  // namespace caffe
