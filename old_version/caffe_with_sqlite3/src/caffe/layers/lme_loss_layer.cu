#include "hip/hip_runtime.h"
/**
 * this is main loss 
 * for category explansion 
 * pair-wise
 **/

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/lme_loss_layer.hpp"

namespace caffe {


//computing category loss pair-wise + unary Distance
template <typename Dtype>
__global__ void LmeLossForwardGPU(const int nthreads,
          const Dtype* bottom_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, Dtype margin, Dtype sigma) {
 CUDA_KERNEL_LOOP(index, nthreads) {
  const int n = index / dim;    //row
  const int label_value = static_cast<int>(label[n]);
  const int label_index = n*dim + label_value;
  // unary version of loss 
  if( label_index == index ) {
   loss[index] += max( Dtype(0),
    bottom_data[index] - sigma);
  } else if( label_index != index ) {
   loss[index] =+ max( Dtype(0),
    margin - bottom_data[label_index] + bottom_data[index] );
   loss[index] += max( Dtype(0),
    sigma - bottom_data[index]);
  }
 }
}

template <typename Dtype>
void LmeLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Dtype closs,wloss;
  wloss = Dtype(0.);
  //computing category loss 
  Blob<Dtype> loss_data(bottom[0]->shape());
  Dtype* loss_data_ = loss_data.mutable_gpu_data();
  const Dtype* score = bottom[0]->gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int num = bottom[0]->num();
  const int dim = bottom[0]->count()/num;
  const int nthreads = num * dim; 
  LmeLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, score, label, loss_data_,
      num, dim , margin,sigma);  
  caffe_gpu_asum(nthreads, loss_data_, &closs);
  wloss+=closs;
  wloss/=num;
  top[0]->mutable_cpu_data()[0] = wloss;
}


template <typename Dtype>
__global__ void LmeLossBackwardGPU( const int nthreads, const Dtype* bottom_data,const Dtype* label, 
                    Dtype* bottom_diff, const int num, const int dim,const Dtype margin,const Dtype sigma) {
 unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
 if(ix < nthreads) { 
  const int label_value = static_cast<int>(label[ix]); 
  const int label_index = dim*ix + label_value;
  for( int iy = 0 ; iy < dim ; iy++){
   const int index = dim*ix + iy;
   if( label_index != index ) {
    //compute pair_wise term
    Dtype prob = max( Dtype(0),
	margin - bottom_data[label_index] + bottom_data[index] );
    if( prob > 0){
       bottom_diff[index] += 1;
       bottom_diff[label_index] -= 1; 
    }
    //compute unary term
    prob = max( Dtype(0),
      sigma - bottom_data[index]);
    if(prob > 0) 
      bottom_diff[index] -= 1;
   } else if ( label_index == index ) {
     Dtype prob = max( Dtype(0),
         bottom_data[index] - sigma);
    if(prob > 0) 
       bottom_diff[index] += 1;
   }        
  }
 }
}

template <typename Dtype>
void LmeLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const int num = bottom[0]->num();
  const Dtype scale = top[0]->cpu_diff()[0]/num;
  //const Dtype scale = top[0]->cpu_diff()[0]/50;
  //const Dtype scale = 1.0/num;
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    const Dtype* score = bottom[0]->gpu_data();
    const Dtype* label = bottom[1]->gpu_data();
    const int dim = bottom[0]->count()/num;
    const int nthreads = num;
    //initialize 
    caffe_gpu_set(bottom[0]->count(),Dtype(0.),bottom[0]->mutable_gpu_diff());
    LmeLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, score, label, bottom[0]->mutable_gpu_diff(),
        num, dim , margin,sigma );
    caffe_gpu_scal(bottom[0]->count(), scale , bottom[0]->mutable_gpu_diff());
 }
}

INSTANTIATE_LAYER_GPU_FUNCS(LmeLossLayer);

}  // namespace caffe
